#include "hip/hip_runtime.h"
// This is a personal academic project. Dear PVS-Studio, please check it.
// PVS-Studio Static Code Analyzer for C, C++ and C#: http://www.viva64.com
#include "coloringMCMC.h"

template<typename nodeW, typename edgeW>
ColoringMCMC<nodeW, edgeW>::ColoringMCMC(Graph<nodeW, edgeW> * inGraph_d, hiprandState * randStates, ColoringMCMCParams param) :
	Colorer<nodeW, edgeW>(inGraph_d),
	graphStruct_d(inGraph_d->getStruct()),
	nnodes(inGraph_d->getStruct()->nNodes),
	prob(inGraph_d->prob),
	randStates(randStates),
	numOfColors(0),
	threadId(0),
	param(param) {

	// configuro la griglia e i blocchi
	numThreads = 64;
	threadsPerBlock = dim3(numThreads, 1, 1);
	blocksPerGrid = dim3((nnodes + threadsPerBlock.x - 1) / threadsPerBlock.x, 1, 1);
	blocksPerGrid_nCol = dim3((param.nCol + threadsPerBlock.x - 1) / threadsPerBlock.x, 1, 1);
	blocksPerGrid_half = dim3(((nnodes / 2) + threadsPerBlock.x - 1) / threadsPerBlock.x, 1, 1);

	// __printMemAlloc();

	cuSts = hipMalloc((void**)&coloring_d, nnodes * sizeof(uint32_t));		cudaCheck(cuSts, __FILE__, __LINE__);
	cuSts = hipMalloc((void**)&starColoring_d, nnodes * sizeof(uint32_t));	cudaCheck(cuSts, __FILE__, __LINE__);

#ifdef TABOO
	cuSts = hipMalloc((void**)&taboo_d, nnodes * sizeof(uint32_t));	cudaCheck(cuSts, __FILE__, __LINE__);
#endif // TABOO

	q_h = (float *)malloc(nnodes * sizeof(float));
	cuSts = hipMalloc((void**)&q_d, nnodes * sizeof(float));	cudaCheck(cuSts, __FILE__, __LINE__);
	qStar_h = (float *)malloc(nnodes * sizeof(float));
	cuSts = hipMalloc((void**)&qStar_d, nnodes * sizeof(float));	cudaCheck(cuSts, __FILE__, __LINE__);

	conflictCounter_h = (uint32_t *)malloc(nnodes * sizeof(uint32_t));
	cuSts = hipMalloc((void**)&conflictCounter_d, nnodes * sizeof(uint32_t));	cudaCheck(cuSts, __FILE__, __LINE__);
	cuSts = hipMalloc((void**)&colorsChecker_d, nnodes * param.nCol * sizeof(bool));	cudaCheck(cuSts, __FILE__, __LINE__);
#if defined(DISTRIBUTION_LINE_INIT) || defined(COLOR_DECREASE_LINE) || defined(COLOR_BALANCE_LINE)
	cuSts = hipMalloc((void**)&probDistributionLine_d, param.nCol * sizeof(float));	cudaCheck(cuSts, __FILE__, __LINE__);
#endif // DISTRIBUTION_LINE_INIT || COLOR_DECREASE_LINE
#if defined(DISTRIBUTION_EXP_INIT) || defined(COLOR_DECREASE_EXP) || defined(COLOR_BALANCE_EXP)
	cuSts = hipMalloc((void**)&probDistributionExp_d, param.nCol * sizeof(float));	cudaCheck(cuSts, __FILE__, __LINE__);
#endif // DISTRIBUTION_EXP_INIT || COLOR_DECREASE_EXP || COLOR_BALANCE_EXP
#ifdef COLOR_BALANCE_DYNAMIC_DISTR
	cuSts = hipMalloc((void**)&probDistributionDynamic_d, param.nCol * sizeof(float));	cudaCheck(cuSts, __FILE__, __LINE__);
#endif // COLOR_BALANCE_DYNAMIC_DISTR

#if defined(COLOR_BALANCE_LINE) || defined(COLOR_BALANCE_EXP) || defined(COLOR_BALANCE_DYNAMIC_DISTR)
	orderedIndex_h = (uint32_t *)malloc(param.nCol * sizeof(uint32_t));
	cuSts = hipMalloc((void**)&orderedIndex_d, param.nCol * sizeof(uint32_t));	cudaCheck(cuSts, __FILE__, __LINE__);
#endif // COLOR_BALANCE_EXP

	coloring_h = (uint32_t *)malloc(nnodes * sizeof(uint32_t));
	statsColors_h = conflictCounter_h;
	statsColors_d = conflictCounter_d;
	statsFreeColors_d = conflictCounter_d;

}

template<typename nodeW, typename edgeW>
ColoringMCMC<nodeW, edgeW>::~ColoringMCMC() {
	cuSts = hipFree(coloring_d); 					cudaCheck(cuSts, __FILE__, __LINE__);
	cuSts = hipFree(starColoring_d); 				cudaCheck(cuSts, __FILE__, __LINE__);

#ifdef TABOO
	cuSts = hipFree(taboo_d); 						cudaCheck(cuSts, __FILE__, __LINE__);
#endif // TABOO

	cuSts = hipFree(colorsChecker_d); 				cudaCheck(cuSts, __FILE__, __LINE__);
#if defined(DISTRIBUTION_LINE_INIT) || defined(COLOR_DECREASE_LINE) || defined(COLOR_BALANCE_LINE)
	cuSts = hipFree(probDistributionLine_d); 		cudaCheck(cuSts, __FILE__, __LINE__);
#endif // DISTRIBUTION_LINE_INIT || COLOR_DECREASE_LINE
#if defined(DISTRIBUTION_EXP_INIT) || defined(COLOR_DECREASE_EXP) || defined(COLOR_BALANCE_EXP)
	cuSts = hipFree(probDistributionExp_d); 		cudaCheck(cuSts, __FILE__, __LINE__);
#endif // DISTRIBUTION_EXP_INIT || COLOR_DECREASE_EXP || COLOR_BALANCE_EXP
#ifdef COLOR_BALANCE_DYNAMIC_DISTR
	cuSts = hipFree(probDistributionDynamic_d); 	cudaCheck(cuSts, __FILE__, __LINE__);
#endif // COLOR_BALANCE_DYNAMIC_DISTR

#if defined(COLOR_BALANCE_LINE) || defined(COLOR_BALANCE_EXP) || defined(COLOR_BALANCE_DYNAMIC_DISTR)
	free(orderedIndex_h);
	cuSts = hipFree(orderedIndex_d); 				cudaCheck(cuSts, __FILE__, __LINE__);
#endif // COLOR_BALANCE_EXP

	cuSts = hipFree(conflictCounter_d); 			cudaCheck(cuSts, __FILE__, __LINE__);
	cuSts = hipFree(q_d); 							cudaCheck(cuSts, __FILE__, __LINE__);
	cuSts = hipFree(qStar_d);						cudaCheck(cuSts, __FILE__, __LINE__);

	free(coloring_h);
	free(conflictCounter_h);
	free(q_h);
	free(qStar_h);
}

/**
 * Start the coloring on the graph
 */
template<typename nodeW, typename edgeW>
void ColoringMCMC<nodeW, edgeW>::run(int iteration) {

	rip = 0;

	__customPrintRun0_start(iteration);

	cuSts = hipMemset(coloring_d, 0, nnodes * sizeof(uint32_t)); cudaCheck(cuSts, __FILE__, __LINE__);

#ifdef TABOO
	cuSts = hipMemset(taboo_d, 0, nnodes * sizeof(uint32_t)); cudaCheck(cuSts, __FILE__, __LINE__);
#endif // TABOO

#if defined(DISTRIBUTION_LINE_INIT) || defined(COLOR_DECREASE_LINE) || defined(COLOR_BALANCE_LINE)
	float denomL = 0;
	for (int i = 0; i < param.nCol; i++)
		denomL += (param.nCol - i * param.lambda);
	ColoringMCMC_k::initDistributionLine << < blocksPerGrid_nCol, threadsPerBlock >> > (param.nCol, denomL, param.lambda, probDistributionLine_d);
	hipDeviceSynchronize();
#endif // DISTRIBUTION_LINE_INIT || COLOR_DECREASE_LINE || COLOR_BALANCE_LINE

#if defined(DISTRIBUTION_EXP_INIT) || defined(COLOR_DECREASE_EXP) || defined(COLOR_BALANCE_EXP)
	float denomE = 0;
	for (int i = 0; i < param.nCol; i++)
		denomE += exp(-param.lambda * i);
	ColoringMCMC_k::initDistributionExp << < blocksPerGrid_nCol, threadsPerBlock >> > (param.nCol, denomE, param.lambda, probDistributionExp_d);
	hipDeviceSynchronize();
#endif // DISTRIBUTION_LINE_INIT || COLOR_DECREASE_LINE || COLOR_BALANCE_EXP

#if defined(COLOR_BALANCE_DYNAMIC_DISTR)
	for (uint32_t i = 0; i < param.nCol; i++) orderedIndex_h[i] = i;
	cuSts = hipMemcpy(orderedIndex_d, orderedIndex_h, param.nCol * sizeof(uint32_t), hipMemcpyHostToDevice); cudaCheck(cuSts, __FILE__, __LINE__);
#endif

#ifdef STANDARD_INIT
	ColoringMCMC_k::initColoring << < blocksPerGrid, threadsPerBlock >> > (nnodes, coloring_d, param.nCol, randStates);
#endif // STANDARD_INIT

#ifdef DISTRIBUTION_LINE_INIT
	ColoringMCMC_k::initColoringWithDistribution << < blocksPerGrid, threadsPerBlock >> > (nnodes, coloring_d, param.nCol, probDistributionLine_d, randStates);
#endif // DISTRIBUTION_LINE_INIT

#ifdef DISTRIBUTION_EXP_INIT
	ColoringMCMC_k::initColoringWithDistribution << < blocksPerGrid, threadsPerBlock >> > (nnodes, coloring_d, param.nCol, probDistributionExp_d, randStates);
#endif // DISTRIBUTION_EXP_INIT
	hipDeviceSynchronize();

	//__customPrintRun1_init(); 		// Prints initial coloring. WHO CARES.

	// Setting the tail cutting threshold.
	if (param.tailcut) {
		z = (50 > nnodes / 2000) ? 50 : (nnodes / 2000);
		LOG(TRACE) << TXT_BIYLW << "Tailcut set to " << z << " nodes" << TXT_NORML;
	}
	else {
		z = 0;
		LOG(TRACE) << TXT_BIYLW << "Tailcut disabled (--tailcut)" << TXT_NORML;
	}

	start = std::clock();
	do {
		rip++;

		calcConflicts(conflictCounter, coloring_d);

#ifdef HASTINGS
		calcStdDev(stdDev, coloring_d);
#endif // HASTINGS

		if (conflictCounter <= z)
			break;

		__customPrintRun2_conflicts(false);

		hipMemset(colorsChecker_d, 0, nnodes * param.nCol * sizeof(bool));

#ifdef STANDARD
		ColoringMCMC_k::selectStarColoring << < blocksPerGrid, threadsPerBlock >> > (nnodes, starColoring_d, qStar_d, param.nCol, coloring_d, graphStruct_d->cumulDegs, graphStruct_d->neighs, colorsChecker_d, taboo_d, param.tabooIteration, randStates, param.epsilon, statsFreeColors_d);
		hipDeviceSynchronize();
#endif // STANDARD

#if defined(COLOR_DECREASE_LINE) || defined(COLOR_DECREASE_EXP)
#ifdef COLOR_DECREASE_LINE
		float * probDistribution_d = probDistributionLine_d;
#endif // COLOR_DECREASE_LINE
#ifdef COLOR_DECREASE_EXP
		float * probDistribution_d = probDistributionExp_d;
#endif // COLOR_DECREASE_EXP

		ColoringMCMC_k::selectStarColoringDecrease << < blocksPerGrid, threadsPerBlock >> > (nnodes, starColoring_d, qStar_d, param.nCol, coloring_d, graphStruct_d->cumulDegs, graphStruct_d->neighs, colorsChecker_d, taboo_d, param.tabooIteration, probDistribution_d, randStates, param.lambda, param.epsilon, statsFreeColors_d);
#endif // COLOR_DECREASE_LINE || COLOR_DECREASE_EXP

#if defined(COLOR_BALANCE_LINE) || defined(COLOR_BALANCE_EXP)
		cuSts = hipMemcpy(coloring_h, coloring_d, nnodes * sizeof(uint32_t), hipMemcpyDeviceToHost); cudaCheck(cuSts, __FILE__, __LINE__);
		memset(statsColors_h, 0, nnodes * sizeof(uint32_t));
		for (int i = 0; i < nnodes; i++) statsColors_h[coloring_h[i]]++;
		for (uint32_t i = 0; i < param.nCol; i++) orderedIndex_h[i] = i;
		std::sort(&orderedIndex_h[0], &orderedIndex_h[param.nCol], [&](int i, int j) {return statsColors_h[i] < statsColors_h[j]; });
		cuSts = hipMemcpy(orderedIndex_d, orderedIndex_h, param.nCol * sizeof(uint32_t), hipMemcpyHostToDevice); cudaCheck(cuSts, __FILE__, __LINE__);

#ifdef COLOR_BALANCE_LINE
		float * probDistribution_d = probDistributionLine_d;
#endif // COLOR_DECREASE_LINE
#ifdef COLOR_BALANCE_EXP
		float * probDistribution_d = probDistributionExp_d;
#endif // COLOR_DECREASE_EXP

		ColoringMCMC_k::selectStarColoringBalance << < blocksPerGrid, threadsPerBlock >> > (nnodes, starColoring_d, qStar_d, param.nCol, coloring_d, graphStruct_d->cumulDegs, graphStruct_d->neighs, colorsChecker_d, taboo_d, param.tabooIteration, probDistribution_d, orderedIndex_d, randStates, param.lambda, param.epsilon, statsFreeColors_d);
#endif // COLOR_BALANCE_LINE || COLOR_BALANCE_EXP

#if defined(COLOR_BALANCE_DYNAMIC_DISTR)
		cuSts = hipMemcpy(coloring_h, coloring_d, nnodes * sizeof(uint32_t), hipMemcpyDeviceToHost); cudaCheck(cuSts, __FILE__, __LINE__);
		memset(statsColors_h, 0, nnodes * sizeof(uint32_t));
		for (int i = 0; i < nnodes; i++) statsColors_h[coloring_h[i]]++;
		cuSts = hipMemcpy(statsColors_d, statsColors_h, param.nCol * sizeof(uint32_t), hipMemcpyHostToDevice); cudaCheck(cuSts, __FILE__, __LINE__);

		ColoringMCMC_k::genDynamicDistribution << < blocksPerGrid_nCol, threadsPerBlock >> > (probDistributionDynamic_d, param.nCol, nnodes, statsColors_d);

		ColoringMCMC_k::selectStarColoringBalanceDynamic << < blocksPerGrid, threadsPerBlock >> > (nnodes, starColoring_d, qStar_d, param.nCol, coloring_d, graphStruct_d->cumulDegs, graphStruct_d->neighs, colorsChecker_d, taboo_d, param.tabooIteration, probDistributionDynamic_d, orderedIndex_d, randStates, param.lambda, param.epsilon, statsColors_d);
#endif

		hipDeviceSynchronize();

#ifdef HASTINGS
		hipMemset(colorsChecker_d, 0, nnodes * param.nCol * sizeof(bool));
		ColoringMCMC_k::lookOldColoring << < blocksPerGrid, threadsPerBlock >> > (nnodes, starColoring_d, q_d, param.nCol, coloring_d, graphStruct_d->cumulDegs, graphStruct_d->neighs, colorsChecker_d, param.epsilon);
		hipDeviceSynchronize();
#endif //HASTINGS

		calcConflicts(conflictCounterStar, starColoring_d);

		__customPrintRun3_newConflicts();

#ifdef HASTINGS
		/*calcStdDev(stdDevStar, starColoring_d);

		std::cout << "stdDev " << stdDev << std::endl;
		std::cout << "stdDevStar " << stdDevStar << std::endl;

		float conflPart = (float)(conflictCounterStar - conflictCounter) / float(conflictCounter);
		std::cout << "conflPart " << -conflPart << std::endl;

		float balancePart = (stdDevStar - stdDev) / stdDev;
		std::cout << "balancePart " << -balancePart << std::endl;

		float l = 0.3, g = 1 - l;
		result = -l * conflPart
			- g * balancePart;
		std::cout << "result " << result << std::endl;*/

		//exec the sums of logs of q and qstar
		calcProbs();
		result = -param.lambda * (conflictCounterStar - conflictCounter) + (p - pStar);
		result = exp(result);

		random = ((float)rand() / (float)RAND_MAX);
		std::cout << "random " << random << std::endl;

		__customPrintRun5();

		__customPrintRun6_change();
#endif //HASTINGS

		switchPointer = coloring_d;
		coloring_d = starColoring_d;
		starColoring_d = switchPointer;

		//getStatsNumColors("running_");

	} while (rip < param.maxRip);

	if(param.tailcut) {
		cuSts = hipMemcpy(coloring_h, coloring_d, nnodes * sizeof(uint32_t), hipMemcpyDeviceToHost); cudaCheck(cuSts, __FILE__, __LINE__);
		memset(statsColors_h, 0, nnodes * sizeof(uint32_t));
		for (int i = 0; i < nnodes; i++) statsColors_h[coloring_h[i]]++;
		for (uint32_t i = 0; i < param.nCol; i++) orderedIndex_h[i] = i;
		std::sort(&orderedIndex_h[0], &orderedIndex_h[param.nCol], [&](int i, int j) {return statsColors_h[i] < statsColors_h[j]; });
		cuSts = hipMemcpy(orderedIndex_d, orderedIndex_h, param.nCol * sizeof(uint32_t), hipMemcpyHostToDevice); cudaCheck(cuSts, __FILE__, __LINE__);

		while (conflictCounter > 0) {
			__customPrintRun2_conflicts(true);
			// set conflictCounter_d vector with 1 or 0 to indicate conflicts
			ColoringMCMC_k::conflictCounter << < blocksPerGrid, threadsPerBlock >> > (nnodes, conflictCounter_d, coloring_d, graphStruct_d->cumulDegs, graphStruct_d->neighs);
			// set colorsChecker_d vector values to 0
			hipMemset(colorsChecker_d, 0, nnodes * param.nCol * sizeof(bool));
			// resolve conflicts
			ColoringMCMC_k::tailCutting << < 1, 1 >> > (nnodes, param.nCol, coloring_d, graphStruct_d->cumulDegs, graphStruct_d->neighs, colorsChecker_d, conflictCounter, conflictCounter_d, orderedIndex_d);
			calcConflicts(conflictCounter, coloring_d);
			__customPrintRun3_newConflicts();
		}
	}

	duration = (std::clock() - start) / (double)CLOCKS_PER_SEC;

	if (rip == param.maxRip)
		maxIterReached = true;

	__customPrintRun7_end();
}

//// Questo serve per mantenere le dechiarazioni e definizioni in classi separate
//// E' necessario aggiungere ogni nuova dichiarazione per ogni nuova classe tipizzata usata nel main
template class ColoringMCMC<col, col>;
template class ColoringMCMC<float, float>;
