#include "hip/hip_runtime.h"
// This is a personal academic project. Dear PVS-Studio, please check it.
// PVS-Studio Static Code Analyzer for C, C++ and C#: http://www.viva64.com
#include "coloringMCMC.h"

__global__ void ColoringMCMC_k::initDistributionLine(float nCol, float denom, float lambda, float * probDistributionLine_d) {
	uint32_t idx = threadIdx.x + blockDim.x * blockIdx.x;
	if (idx >= nCol)
		return;

	probDistributionLine_d[idx] = (float)(nCol - lambda * idx) / denom;
	//probDistributionLine_d[idx] = (float)(lambda * idx) / denom;
}


__global__ void ColoringMCMC_k::initDistributionExp(float nCol, float denom, float lambda, float * probDistributionExp_d) {
	uint32_t idx = threadIdx.x + blockDim.x * blockIdx.x;
	if (idx >= nCol)
		return;

	probDistributionExp_d[idx] = exp(-lambda * idx) / denom;
}


__global__ void ColoringMCMC_k::initColoring(uint32_t nnodes, uint32_t * coloring_d, float nCol, hiprandState * states) {
	uint32_t idx = threadIdx.x + blockDim.x * blockIdx.x;
	if (idx >= nnodes)
		return;

	float randnum = hiprand_uniform(&states[idx]);
	int color = (int)(randnum * nCol);
	coloring_d[idx] = color;
	//coloring_d[idx] = 0;
}


__global__ void ColoringMCMC_k::initColoringWithDistribution(uint32_t nnodes, uint32_t * coloring_d, float nCol, float * probDistribution_d, hiprandState * states) {
	uint32_t idx = threadIdx.x + blockDim.x * blockIdx.x;
	if (idx >= nnodes)
		return;

	float randnum = hiprand_uniform(&states[idx]);

	int color = 0;
	float threshold = 0;
	while (threshold < randnum) {
		threshold += probDistribution_d[color];
		color++;
	}

	/*if (idx == 0) {
		float a = 0;
		for (int i = 0; i < nCol; i++)
		{
			a += probDistribution_d[i];
			printf("parziale : %f\n", probDistribution_d[i]);
		}
		printf("totale : %f\n", a);
	}*/

	coloring_d[idx] = color - 1;
}


__global__ void ColoringMCMC_k::genDynamicDistribution(float * probDistributionDynamic_d, uint32_t nCol, uint32_t nnodes, uint32_t * statsColors_d) {
	uint32_t idx = threadIdx.x + blockDim.x * blockIdx.x;
	if (idx >= nCol)
		return;

	probDistributionDynamic_d[idx] = (1 - ((float)statsColors_d[idx] / (float)nnodes)) / (float)(nCol - 1);
}


__global__ void ColoringMCMC_k::tailCutting(uint32_t nnodes, col_sz nCol, uint32_t * coloring_d, node_sz * cumulDegs, node * neighs, bool * colorsChecker_d, int conflictCounter, uint32_t * conflictCounter_d, uint32_t * orderedIndex_d) {
	if (threadIdx.x + blockDim.x * blockIdx.x >= 1)
		return;

	int resolved = 0;
	for (uint32_t idx = 0; idx < nnodes && resolved < conflictCounter; idx++) {
		if (conflictCounter_d[idx]) {
			resolved++;

			uint32_t index = cumulDegs[idx];								//index of the node in neighs
			uint32_t nneighs = cumulDegs[idx + 1] - index;					//number of neighbors

			uint32_t nodeCol = coloring_d[idx];								//node color

			bool * colorsChecker = &(colorsChecker_d[idx * nCol]);			//array used to set to 1 or 0 the colors occupied from the neighbors
			for (int i = 0; i < nneighs; i++)
				colorsChecker[coloring_d[neighs[index + i]]] = 1;

			int j = 0;
			while (colorsChecker[nodeCol] && j < nCol) {
				nodeCol = orderedIndex_d[j];
				j++;
			}

			coloring_d[idx] = nodeCol;
		}
	}

}

__global__ void ColoringMCMC_k::conflictCounter(uint32_t nnodes, uint32_t * conflictCounter_d, uint32_t * coloring_d, node_sz * cumulDegs, node * neighs) {
	uint32_t idx = threadIdx.x + blockDim.x * blockIdx.x;
	if (idx >= nnodes)
		return;

	uint32_t index = cumulDegs[idx];							//index of the node in neighs
	uint32_t nneighs = cumulDegs[idx + 1] - index;				//number of neighbors

	uint32_t nodeCol = coloring_d[idx];							//node color

	uint32_t conflicts = 0;
	for (int i = 0; i < nneighs; i++)
		conflicts += (coloring_d[neighs[index + i]] == nodeCol) && (idx < neighs[index + i]);
	//conflicts += (coloring_d[neighs[index + i]] == nodeCol);

	conflictCounter_d[idx] = conflicts;
}


__device__ void ColoringMCMC_k::warpReduction(volatile float *sdata, uint32_t tid, uint32_t blockSize) {
	if (blockSize >= 64) sdata[tid] += sdata[tid + 32];
	if (blockSize >= 32) sdata[tid] += sdata[tid + 16];
	if (blockSize >= 16) sdata[tid] += sdata[tid + 8];
	if (blockSize >= 8) sdata[tid] += sdata[tid + 4];
	if (blockSize >= 4) sdata[tid] += sdata[tid + 2];
	if (blockSize >= 2) sdata[tid] += sdata[tid + 1];
}

/*
* Parallel sum reduction inside a block and write the partial result in conflictCounter_d.
* At the end, conflictCounter_d have n partial results for the first n positions where n is the number of blocks called.

* refs: https://developer.download.nvidia.com/assets/cuda/files/reduction.pdf
*/
__global__ void ColoringMCMC_k::sumReduction(uint32_t n, float * conflictCounter_d) {
	uint32_t idx = threadIdx.x + blockDim.x * blockIdx.x;
	if (idx >= n)
		return;

	extern	__shared__ float sdata[];

	uint32_t tid = threadIdx.x;
	uint32_t blockSize = blockDim.x;
	uint32_t i = (blockSize * 2) * blockIdx.x + tid;

	sdata[tid] = conflictCounter_d[i] + conflictCounter_d[i + blockSize];

	/*uint32_t gridSize = (blockSize * 2) * gridDim.x;
	sdata[tid] = 0;
	while (i < n) {
		sdata[tid] += conflictCounter_d[i] + conflictCounter_d[i + blockSize];
		i += gridSize;
	}*/
	__syncthreads();

	//useless for blocks of dim <= 64
	if (blockSize >= 512) {
		if (tid < 256)
			sdata[tid] += sdata[tid + 256];
		__syncthreads();
	}
	if (blockSize >= 256) {
		if (tid < 128)
			sdata[tid] += sdata[tid + 128];
		__syncthreads();
	}
	if (blockSize >= 128) {
		if (tid < 64)
			sdata[tid] += sdata[tid + 64];
		__syncthreads();
	}

	if (tid < 32)
		//ColoringMCMC_k::warpReduction<blockSize>(sdata, tid);
		ColoringMCMC_k::warpReduction(sdata, tid, blockSize);

	if (tid == 0)
		conflictCounter_d[blockIdx.x] = sdata[0];
}


template<typename nodeW, typename edgeW>
void ColoringMCMC<nodeW, edgeW>::calcConflicts(int &conflictCounter, uint32_t * coloring_d) {
	ColoringMCMC_k::conflictCounter << < blocksPerGrid, threadsPerBlock >> > (nnodes, conflictCounter_d, coloring_d, graphStruct_d->cumulDegs, graphStruct_d->neighs);
	hipDeviceSynchronize();

	ColoringMCMC_k::sumReduction << < blocksPerGrid_half, threadsPerBlock, threadsPerBlock.x * sizeof(uint32_t) >> > (nnodes, (float*)conflictCounter_d);
	hipDeviceSynchronize();

	cuSts = hipMemcpy(conflictCounter_h, conflictCounter_d, blocksPerGrid_half.x * sizeof(node_sz), hipMemcpyDeviceToHost); cudaCheck(cuSts, __FILE__, __LINE__);

	conflictCounter = 0;

	for (int i = 0; i < blocksPerGrid_half.x; i++)
		conflictCounter += conflictCounter_h[i];
}


__global__ void ColoringMCMC_k::logarithmer(uint32_t nnodes, float * values) {
	uint32_t idx = threadIdx.x + blockDim.x * blockIdx.x;
	if (idx >= nnodes)
		return;

	// TODO: zio, qui si possono usare le istruzioni hw cuda!
	values[idx] = log(values[idx]);
}


template<typename nodeW, typename edgeW>
void ColoringMCMC<nodeW, edgeW>::calcStdDev(float &std, uint32_t * col_d) {
	cuSts = hipMemcpy(coloring_h, col_d, nnodes * sizeof(uint32_t), hipMemcpyDeviceToHost); cudaCheck(cuSts, __FILE__, __LINE__);
	memset(statsColors_h, 0, nnodes * sizeof(uint32_t));
	for (int i = 0; i < nnodes; i++) {
		statsColors_h[coloring_h[i]]++;
	}

	float average = (float)nnodes / param.nCol;
	float variance = 0;

	for (int i = 0; i < param.nCol; i++) {
		variance += pow((statsColors_h[i] - average), 2.f);
	}
	variance /= param.nCol;

	std = sqrt(variance);
}


template<typename nodeW, typename edgeW>
void ColoringMCMC<nodeW, edgeW>::calcProbs() {

	ColoringMCMC_k::logarithmer << < blocksPerGrid, threadsPerBlock >> > (nnodes, qStar_d);
	ColoringMCMC_k::logarithmer << < blocksPerGrid, threadsPerBlock >> > (nnodes, q_d);
	hipDeviceSynchronize();

	ColoringMCMC_k::sumReduction << < blocksPerGrid_half, threadsPerBlock, threadsPerBlock.x * sizeof(float) >> > (nnodes, qStar_d);
	ColoringMCMC_k::sumReduction << < blocksPerGrid_half, threadsPerBlock, threadsPerBlock.x * sizeof(float) >> > (nnodes, q_d);
	hipDeviceSynchronize();

	cuSts = hipMemcpy(qStar_h, qStar_d, blocksPerGrid_half.x * sizeof(float), hipMemcpyDeviceToHost); cudaCheck(cuSts, __FILE__, __LINE__);
	cuSts = hipMemcpy(q_h, q_d, blocksPerGrid_half.x * sizeof(float), hipMemcpyDeviceToHost); cudaCheck(cuSts, __FILE__, __LINE__);

	pStar = 0;
	p = 0;
	for (int i = 0; i < blocksPerGrid_half.x; i++) {
		pStar += qStar_h[i];
		p += q_h[i];
	}
}

//// Questo serve per mantenere le dechiarazioni e definizioni in classi separate
//// E' necessario aggiungere ogni nuova dichiarazione per ogni nuova classe tipizzata usata nel main
template class ColoringMCMC<col, col>;
template class ColoringMCMC<float, float>;
