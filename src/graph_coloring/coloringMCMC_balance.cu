#include "hip/hip_runtime.h"
// This is a personal academic project. Dear PVS-Studio, please check it.
// PVS-Studio Static Code Analyzer for C, C++ and C#: http://www.viva64.com
#include "coloringMCMC.h"

__global__ void ColoringMCMC_k::selectStarColoringBalance(uint32_t nnodes, uint32_t * starColoring_d, float * qStar_d, col_sz nCol, uint32_t * coloring_d, node_sz * cumulDegs, node * neighs, bool * colorsChecker_d, uint32_t * taboo_d, uint32_t tabooIteration, float * probDistribution_d, uint32_t * orderedIndex_d, hiprandState * states, float lambda, float epsilon, uint32_t * statsFreeColors_d) {
	uint32_t idx = threadIdx.x + blockDim.x * blockIdx.x;
	if (idx >= nnodes)
		return;

#ifdef TABOO
	if (taboo_d[idx] > 0) {
		taboo_d[idx]--;
		qStar_d[idx] = (1.0f - (nCol - 1) * epsilon);			//save the probability of the color chosen
		return;
	}
#endif // TABOO

	uint32_t index = cumulDegs[idx];							//index of the node in neighs
	uint32_t nneighs = cumulDegs[idx + 1] - index;				//number of neighbors
	uint32_t nodeCol = coloring_d[idx];							//node color

	bool * colorsChecker = &(colorsChecker_d[idx * nCol]);		//array used to set if a color is used from the neighbors
	for (int i = 0; i < nneighs; i++) {
		colorsChecker[coloring_d[neighs[index + i]]] = 1;
	}

	float reminder = 0;
	uint32_t Zn = 0, Zp = nCol;									//number of free colors (p) and occupied colors (n)
	for (int i = 0; i < nCol; i++) {
		Zn += colorsChecker[i];
		reminder += colorsChecker[i] * (probDistribution_d[orderedIndex_d[i]] - epsilon);
	}
	Zp -= Zn;

	if (!Zp) {													//manage exception of no free colors
		starColoring_d[idx] = nodeCol;
		qStar_d[idx] = 1;
		return;
	}

	float denomReminder = 0;
	//for (int i = 0; i < Zp; i++)
	//{
	//	denomReminder += exp(-lambda * i);
	//}
	denomReminder = Zp;

	int i = 0;
	//int j = 0;
	float q;
	float threshold = 0;
	float randnum = hiprand_uniform(&states[idx]);				//random number
	if (colorsChecker[nodeCol]) {								//if node color is used by neighbors
		do {
			//float r = reminder * (exp(-lambda * j) / denomReminder);
			float r = reminder / denomReminder;
			q = (probDistribution_d[orderedIndex_d[i]] + r) * (!colorsChecker[i]) + (epsilon) * (colorsChecker[i]);
			threshold += q;
			//j += !colorsChecker[i];
			i++;
		} while (threshold < randnum && i < nCol);
	}
	else {
		do {
			q = (1.0f - (nCol - 1) * epsilon) * (nodeCol == i) + (epsilon) * (nodeCol != i);
			threshold += q;
			i++;
		} while (threshold < randnum && i < nCol);
	}
	qStar_d[idx] = q;											//save the probability of the color chosen
	starColoring_d[idx] = i - 1;

#ifdef TABOO
	taboo_d[idx] = (starColoring_d[idx] == nodeCol) * tabooIteration;
#endif // TABOO
}


__global__ void ColoringMCMC_k::selectStarColoringBalanceDynamic(uint32_t nnodes, uint32_t * starColoring_d, float * qStar_d, col_sz nCol, uint32_t * coloring_d, node_sz * cumulDegs, node * neighs, bool * colorsChecker_d, uint32_t * taboo_d, uint32_t tabooIteration, float * probDistribution_d, uint32_t * orderedIndex_d, hiprandState * states, float lambda, float epsilon, uint32_t * statsColors_d) {
	uint32_t idx = threadIdx.x + blockDim.x * blockIdx.x;
	if (idx >= nnodes)
		return;

#ifdef TABOO
	if (taboo_d[idx] > 0) {
		taboo_d[idx]--;
		qStar_d[idx] = (1.0f - (nCol - 1) * epsilon);			//save the probability of the color chosen
		return;
	}
#endif // TABOO

	uint32_t index = cumulDegs[idx];							//index of the node in neighs
	uint32_t nneighs = cumulDegs[idx + 1] - index;				//number of neighbors
	uint32_t nodeCol = coloring_d[idx];							//node color

	bool * colorsChecker = &(colorsChecker_d[idx * nCol]);		//array used to set if a color is used from the neighbors
	for (int i = 0; i < nneighs; i++) {
		colorsChecker[coloring_d[neighs[index + i]]] = 1;
	}

	float reminder = 0;
	float denomReminder = 0;
	uint32_t Zn = 0, Zp = nCol;									//number of free colors (p) and occupied colors (n)
	for (int i = 0; i < nCol; i++) {
		Zn += colorsChecker[i];
		reminder += colorsChecker[i] * (probDistribution_d[orderedIndex_d[i]] - epsilon);
	}
	Zp -= Zn;
	denomReminder = Zp;

	if (!Zp) {													//manage exception of no free colors
		starColoring_d[idx] = nodeCol;
		qStar_d[idx] = 1;
		return;
	}

	int i = 0;
	float q;
	float threshold = 0;
	float randnum = hiprand_uniform(&states[idx]);				//random number

	if (colorsChecker[nodeCol]) {								//if node color is used by neighbors
		do {
			float r = reminder / denomReminder;
			q = (probDistribution_d[orderedIndex_d[i]] + r) * (!colorsChecker[i]) + (epsilon) * (colorsChecker[i]);
			threshold += q;
			i++;
		} while (threshold < randnum && i < nCol);
	}
	else {
		do {
			q = (1.0f - (nCol - 1) * epsilon) * (nodeCol == i) + (epsilon) * (nodeCol != i);
			threshold += q;
			i++;
		} while (threshold < randnum && i < nCol);
	}
	qStar_d[idx] = q;											//save the probability of the color chosen
	starColoring_d[idx] = i - 1;

#ifdef TABOO
	taboo_d[idx] = (starColoring_d[idx] == nodeCol) * tabooIteration;
#endif // TABOO
}
